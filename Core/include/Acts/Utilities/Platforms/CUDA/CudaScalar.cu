#pragma once

#include <iostream>
#include <memory>
#include "hip/hip_runtime.h"


namespace Acts{

template<typename Var_t>
class CudaScalar{

public:
  CudaScalar(){
    hipMalloc((Var_t**)&fDevPtr, sizeof(Var_t));  
  }

  CudaScalar(Var_t* scalar){
    hipMalloc((Var_t**)&fDevPtr, sizeof(Var_t));
    hipMemcpy(fDevPtr, scalar, sizeof(Var_t), hipMemcpyHostToDevice);
  }

  CudaScalar(const Var_t* scalar){
    hipMalloc((Var_t**)&fDevPtr, sizeof(Var_t));
    hipMemcpy(fDevPtr, scalar, sizeof(Var_t), hipMemcpyHostToDevice);
  }
  
  ~CudaScalar(){ 
    hipFree(fDevPtr); 
  }

  Var_t* Get() { return fDevPtr; }
  
  private:
  Var_t* fDevPtr;  
};
}
