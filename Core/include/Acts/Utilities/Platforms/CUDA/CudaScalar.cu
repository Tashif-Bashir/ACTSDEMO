#pragma once

#include <iostream>
#include <memory>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "CudaUtils.cu"

namespace Acts{

template<typename Var_t>
class CudaScalar{

public:
  CudaScalar(){
    cudaErrChk( hipMalloc((Var_t**)&fDevPtr, sizeof(Var_t)) );
  }

  CudaScalar(Var_t* scalar){
    cudaErrChk( hipMalloc((Var_t**)&fDevPtr, sizeof(Var_t)) );
    cudaErrChk( hipMemcpy(fDevPtr, scalar, sizeof(Var_t), hipMemcpyHostToDevice) );
  }

  CudaScalar(const Var_t* scalar){
    cudaErrChk( hipMalloc((Var_t**)&fDevPtr, sizeof(Var_t)) );
    cudaErrChk( hipMemcpy(fDevPtr, scalar, sizeof(Var_t), hipMemcpyHostToDevice) );
  }
  
  ~CudaScalar(){ 
    cudaErrChk( hipFree(fDevPtr) );
  }

  Var_t* Get() { return fDevPtr; }
  
  private:
  Var_t* fDevPtr;  
};
}
