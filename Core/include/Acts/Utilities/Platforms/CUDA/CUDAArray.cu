#pragma once

#include <iostream>
#include <memory>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "Acts/Utilities/Platforms/CUDA/CPUArray.hpp"

namespace Acts{

template<typename Var_t>
class CUDAArray{

public:
  
  CUDAArray(size_t size){ 
    fSize = size;
    hipMalloc((Var_t**)&fDevPtr, fSize*sizeof(Var_t));
  }

  CUDAArray(size_t size, Var_t* buffer, size_t len, size_t offset=0){ 
    fSize = size;
    hipMalloc((Var_t**)&fDevPtr, fSize*sizeof(Var_t));
    CopyH2D(buffer, len, offset);
  }

  CUDAArray(size_t size, const Var_t* buffer, size_t len, size_t offset=0){ 
    fSize = size;
    hipMalloc((Var_t**)&fDevPtr, fSize*sizeof(Var_t));
    CopyH2D(buffer, len, offset);
  }
  
  ~CUDAArray(){ 
    hipFree(fDevPtr); 
  }

  size_t GetSize(){return fSize;}
  
  Var_t* Get(size_t offset=0) { return fDevPtr+offset; }
  
  Var_t* GetHostArray(size_t len, size_t offset=0) const {
    Var_t* hostArray = new Var_t[len];
    hipMemcpy(hostArray, fDevPtr+offset, len*sizeof(Var_t), hipMemcpyDeviceToHost);   
    return hostArray;
  }

  CPUArray<Var_t>* GetCPUArray(size_t len, size_t offset=0) const {
    CPUArray<Var_t>* cpuArray = new CPUArray<Var_t>(len);
    hipMemcpy(cpuArray->Get(), fDevPtr+offset, len*sizeof(Var_t), hipMemcpyDeviceToHost);   
    return cpuArray;
  }
  
  //Var_t& operator[](std::size_t idx)       { return fDevPtr[idx]; }  // Need to test
  //const Var_t& operator[](std::size_t idx) const { return fDevPtr[idx]; }  // Need to test
  
  void CopyH2D(Var_t* array, size_t len, size_t offset=0){
    hipMemcpy(fDevPtr+offset, array, len*sizeof(Var_t), hipMemcpyHostToDevice);
  }

  void CopyH2D(const Var_t* array, size_t len, size_t offset=0){
    hipMemcpy(fDevPtr+offset, array, len*sizeof(Var_t), hipMemcpyHostToDevice);
  }
    
private:
  Var_t* fDevPtr; 
  size_t fSize;
};
}
