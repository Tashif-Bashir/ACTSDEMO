#pragma once

#include <hip/hip_runtime.h>

#include <iostream>

#define cudaErrChk(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr,"CUDAassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}
