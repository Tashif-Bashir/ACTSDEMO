#pragma once

#include <iostream>
#include <memory>
#include "hip/hip_runtime.h"


namespace Acts{

template<typename Var_t>
class CudaVector{

public:
  
  CudaVector(size_t size){ 
    fSize = size;
    hipMalloc((Var_t**)&fDevPtr, fSize*sizeof(Var_t));
  }

  CudaVector(size_t size, Var_t* vector){
    fSize = size;
    hipMalloc((Var_t**)&fDevPtr, fSize*sizeof(Var_t));
    CopyH2D(vector, fSize, 0);
  }
     
  CudaVector(size_t size, Var_t* vector, size_t len, size_t offset){ 
    fSize = size;
    hipMalloc((Var_t**)&fDevPtr, fSize*sizeof(Var_t));
    CopyH2D(vector, len, offset);
  }
  
  ~CudaVector(){ 
    hipFree(fDevPtr); 
  }

  size_t GetSize(){return fSize;}
  
  Var_t* Get(size_t offset=0) { return fDevPtr+offset; }

  Var_t* GetHost() {
    Var_t* fHostPtr = new Var_t[fSize];
    hipMemcpy(fHostPtr, fDevPtr, fSize*sizeof(Var_t), hipMemcpyDeviceToHost);
    return fHostPtr;
  }

  void CopyH2D(Var_t* vector, size_t len, size_t offset){
    hipMemcpy(fDevPtr+offset, vector, len*sizeof(Var_t), hipMemcpyHostToDevice);
  }
  void CopyH2D(Var_t* vector, size_t len, size_t offset, hipStream_t* stream){
    hipMemcpyAsync(fDevPtr+offset, vector, len*sizeof(Var_t), hipMemcpyHostToDevice, *stream);
  }
      
private:
  Var_t* fDevPtr; 
  size_t fSize;
};
}
