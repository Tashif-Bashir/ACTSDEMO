#pragma once

#include "Acts/Utilities/Platforms/CUDA/CUDAArray.cu"
#include "Acts/Utilities/Platforms/CUDA/CPUMatrix.hpp"

namespace Acts{

template<typename Var_t>
class CUDAMatrix{

public:

  CUDAMatrix()=default;
  CUDAMatrix(size_t nRows, size_t nCols){
    fNRows = nRows;
    fNCols = nCols;
    hipMalloc((Var_t**)&fDevPtr, fNRows*fNCols*sizeof(Var_t));
  }

  CUDAMatrix(size_t nRows, size_t nCols, CPUMatrix<Var_t>* mat){
    fNRows = nRows;
    fNCols = nCols;
    hipMalloc((Var_t**)&fDevPtr, fNRows*fNCols*sizeof(Var_t));
    CopyH2D(mat->GetEl(0,0),fNRows*fNCols,0);
  }
  
  ~CUDAMatrix(){
    hipFree(fDevPtr);
  }

  size_t GetNCols(){ return fNCols; }
  size_t GetNRows(){ return fNRows; }

  Var_t* GetEl(size_t row, size_t col){
    int offset = row+col*fNRows;
    return fDevPtr+offset;
  }

  CPUArray<Var_t>* GetCPUArray(size_t len, size_t row, size_t col){
    int offset = row+col*fNRows;
    CPUArray<Var_t>* cpuArray = new CPUArray<Var_t>(len);
    hipMemcpy(cpuArray->Get(), fDevPtr+offset, len*sizeof(Var_t), hipMemcpyDeviceToHost);   
    return cpuArray;
  }
  
  void CopyH2D(Var_t* array, size_t len, size_t offset=0){
    hipMemcpy(fDevPtr+offset, array, len*sizeof(Var_t), hipMemcpyHostToDevice);
  }

  void CopyH2D(const Var_t* array, size_t len, size_t offset=0){
    hipMemcpy(fDevPtr+offset, array, len*sizeof(Var_t), hipMemcpyHostToDevice);
  }

private:
  Var_t* fDevPtr; 
  size_t fNCols;
  size_t fNRows;
};

}

