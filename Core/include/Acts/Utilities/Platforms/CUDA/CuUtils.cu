#pragma once

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#define WARP_SIZE 32
#define MAX_BLOCK_SIZE 1024

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess)
    {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
    }
}


void SetDevice(int devID, std::string& devName){
  hipDeviceReset();
  
  hipError_t error;
  hipDeviceProp_t deviceProp;
  error = hipSetDevice(devID);
  error = hipGetDevice(&devID);
  
  if (error != hipSuccess){
    printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
  }
  error = hipGetDeviceProperties(&deviceProp, devID);
  
  if (deviceProp.computeMode == hipComputeModeProhibited){
    fprintf(stderr, "Error: device is running in <Compute Mode Prohibited>, no threads can use ::hipSetDevice().\n");
    exit(EXIT_SUCCESS);
  }
  
  if (error != hipSuccess)  {
    printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
  }
  else{
    printf("\n GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
  }
  
  devName = deviceProp.name;
}
